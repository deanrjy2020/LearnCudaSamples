#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also
// illustrates how to introduce dependencies between CUDA streams with the
// hipStreamWaitEvent function.
//

// Devices of compute capability 2.0 or higher can overlap the kernels
//
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// This is a kernel that does no real work but runs at least for a specified
// number of clocks
__global__ void clock_block(clock_t *d_o, clock_t clock_count) {
  unsigned int start_clock = (unsigned int)clock();

  clock_t clock_offset = 0;

  // 运行够clock_count个时钟周期, 即这么多ticks就可以退出了.
  // 核函数的output就是把这个clock_offset (应该略大于clock_count)输出
  while (clock_offset < clock_count) {
    unsigned int end_clock = (unsigned int)clock();

    // The code below should work like
    // this (thanks to modular arithmetics):
    //
    // clock_offset = (clock_t) (end_clock > start_clock ?
    //                           end_clock - start_clock :
    //                           end_clock + (0xffffffffu - start_clock));
    //
    // Indeed, let m = 2^32 then
    // end - start = end + m - start (mod m).

    clock_offset = (clock_t)(end_clock - start_clock);
  }

  d_o[0] = clock_offset;
}

// Single warp reduction kernel
// N=8, d_clocks就是一个长度为8个array, 放着8个clocks, 即前面8个kernel的执行时间, 单位tick
// 把这些时间加总起来，写回 d_clocks[0]
__global__ void sum(clock_t *d_clocks, int N) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // 分配了一个大小为 32 的共享内存数组，用于归约中间结果
  // 这里假设最多只有 32 个线程（即一个 warp），因为这个是 single-warp reduction
  __shared__ clock_t s_clocks[32];

  clock_t my_sum = 0;

  // thread 0 处理 i=0, i=32, i=64
  // N=8, 只有thread 0~7在工作, 从global mem读入, 写到自己的register my_sum里面(单单copy).
  // 其他thread无用(对应的my_sum=0).
  for (int i = threadIdx.x; i < N; i += blockDim.x) {
    my_sum += d_clocks[i];
  }

  // 每个thread把自己的my_sum写到对应的smem位置.
  s_clocks[threadIdx.x] = my_sum;
  // 等待所有线程完成写入
  cg::sync(cta);

  // 归约（Reduction）逻辑
  // 在smem里面完成规约.
  // 对0~31共32个thread, i=16, 8, 4, 2, 1
  for (int i = 16; i > 0; i /= 2) {
    // i=16, 0~15号thread才能进来, 16~31号thread跳过.
    if (threadIdx.x < i) {
      // 0~15号thread的i=16 (第一次loop), 把smem的后半部分加到前半部分.
      s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
    }

    // 每次都是32个thread都做完了在进行下一次loop
    cg::sync(cta);

    // loop2, i=8, 32个thread都有这个loop, 但是只有0~7号thread才能进if
    //    进去后把4~7号的4个加到0~3上来
    // loop3, i=4, 32个thread都有这个loop, 但是只有0~3号thread才能进if
    //    进去后把2~3号的2个加到0~1上来
    // loop4, i=2, 32个thread都有这个loop, 但是只有0~1号thread才能进if
    //    进去后把1号的1个加到0上来
    // 最终32个smem里面的累加在s_clock[0]上
  }

  // 从smem写回gmem
  d_clocks[0] = s_clocks[0];
}

int main(int argc, char **argv) {
  int nkernels = 8;             // number of concurrent kernels
  int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
  int nbytes = nkernels * sizeof(clock_t);  // number of data bytes
  // 让每个kernel跑10ms
  float kernel_time = 10;                   // time the kernel should run in ms
  float elapsed_time;                       // timing variables
  int cuda_device = 0;

  printf("[%s] - Starting...\n", argv[0]);

  // get number of kernels if overridden on the command line
  if (checkCmdLineFlag(argc, (const char **)argv, "nkernels")) {
    nkernels = getCmdLineArgumentInt(argc, (const char **)argv, "nkernels");
    nstreams = nkernels + 1;
  }

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  cuda_device = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDevice(&cuda_device));

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

  if ((deviceProp.concurrentKernels == 0)) {
    printf("> GPU does not support concurrent kernel execution\n");
    printf("  CUDA kernel runs will be serialized\n");
  }

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

  // allocate host memory
  clock_t *a = 0;  // pointer to the array data in host memory
  checkCudaErrors(hipHostMalloc((void **)&a, nbytes));

  // allocate device memory
  clock_t *d_a = 0;  // pointers to data and init value in the device memory
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));

  // allocate and initialize an array of stream handles
  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  for (int i = 0; i < nstreams; i++) {
    checkCudaErrors(hipStreamCreate(&(streams[i])));
  }

  // create CUDA event handles
  // 用来测量总的 GPU 执行时间, 被记录在stream 0上
  hipEvent_t start_event, stop_event;
  checkCudaErrors(hipEventCreate(&start_event));
  checkCudaErrors(hipEventCreate(&stop_event));

  // the events are used for synchronization only and hence do not need to
  // record timings this also makes events not introduce global sync points when
  // recorded which is critical to get overlap
  // 8个kernel对应8个event
  hipEvent_t *kernelEvent;
  kernelEvent = (hipEvent_t *)malloc(nkernels * sizeof(hipEvent_t));

  // 使用 hipEventDisableTiming 表示这些事件只是用于同步，不记录时间（避免引入隐式同步，影响并发性）
  for (int i = 0; i < nkernels; i++) {
    checkCudaErrors(
        hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming));
  }

  //////////////////////////////////////////////////////////////////////
  // time execution with nkernels streams
  // kernel_time 是期望 kernel 运行的毫秒数, 10ms
  // deviceProp.clockRate 是 GPU 的时钟频率（单位 kHz）, 即1377000 kHZ, 是每秒有多少个时钟周期,
  // 即1377000000 ticks/s
  // 两个相乘表示运行10ms需要的ticks个数 = 10ms * 1/1000 * 1377000000 = 10 * 1377000 = 13770000 ticks
  // 这个就是核函数里面的time_count.
  clock_t total_clocks = 0;
#if defined(__arm__) || defined(__aarch64__)
  // the kernel takes more time than the channel reset time on arm archs, so to
  // prevent hangs reduce time_clocks.
  // 这里arm架构改成和x86架构一样, 因为后面的打印信息没考虑到这个100.
  //clock_t time_clocks = (clock_t)(kernel_time * (deviceProp.clockRate / 100));
  clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
#else
  clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
#endif

  hipEventRecord(start_event, 0);

  // queue nkernels in separate streams and record when they are done
  for (int i = 0; i < nkernels; ++i) {
    // 8个核函数在8个stream上跑, 并行, 一个kernel只有一个thread
    clock_block<<<1, 1, 0, streams[i]>>>(&d_a[i], time_clocks);
    total_clocks += time_clocks;
    // 在每个核函数后面都放一个event
    checkCudaErrors(hipEventRecord(kernelEvent[i], streams[i]));

    // make the last stream wait for the kernel event to be recorded
    // stream 9个, 0~8, kernel 8个, 0~7
    // 最后一个stream (stream 8)等着前面0~7个kernel, 它会等待所有 kernel 执行完再继续执行汇总操作
    checkCudaErrors(
        hipStreamWaitEvent(streams[nstreams - 1], kernelEvent[i], 0));
  }

  // queue a sum kernel and a copy back to host in the last stream.
  // the commands in this stream get dispatched as soon as all the kernel events
  // have been recorded
  // 32个thread, sum kernel 负责汇总多个 d_a[i] 的数据, 然后用异步拷贝从 device 到 host.
  // 两个操作都在 streams 8上执行，而这个 stream 8 又等待了所有 kernel event，
  // 因此它们会在所有 kernel 执行完之后才开始.
  sum<<<1, 32, 0, streams[nstreams - 1]>>>(d_a, nkernels);
  checkCudaErrors(hipMemcpyAsync(
      a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams - 1]));

  // at this point the CPU has dispatched all work for the GPU and can continue
  // processing other tasks in parallel

  // in this sample we just wait until the GPU is done
  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

  // expected 信息, 每个kernel跑10ms, 共8个, 80ms=0.08s
  printf("Expected time for serial execution of %d kernels = %.3fs\n", nkernels,
         nkernels * kernel_time / 1000.0f);
  // 理论完全并行, 8个和1个时间一样, 0.01s
  printf("Expected time for concurrent execution of %d kernels = %.3fs\n",
         nkernels, kernel_time / 1000.0f);
  printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

  bool bTestResult = (a[0] > total_clocks);
  printf("Measured tick = %d, total_clocks = %d\n",
    (unsigned int) a[0], (unsigned int)total_clocks);

  // release resources
  for (int i = 0; i < nkernels; i++) {
    hipStreamDestroy(streams[i]);
    hipEventDestroy(kernelEvent[i]);
  }

  free(streams);
  free(kernelEvent);

  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);
  hipHostFree(a);
  hipFree(d_a);

  if (!bTestResult) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}

/*

[./concurrentKernels] - Starting...
GPU Device 0: "Xavier" with compute capability 7.2

> Detected Compute SM 7.2 hardware with 8 multi-processors
Expected time for serial execution of 8 kernels = 0.080s
Expected time for concurrent execution of 8 kernels = 0.010s
Measured time for sample = 0.022s
Measured tick = 110160137, total_clocks = 110160000
Test passed

*/