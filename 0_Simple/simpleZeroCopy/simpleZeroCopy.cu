#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif


/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

// Allocate generic memory with malloc() and pin it laster instead of using hipHostAlloc()
bool bPinGenericMemory = false;

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

int main(int argc, char **argv)
{
    int n, nelem, deviceCount;
    int idev = 0; // use default device 0
    char *device = NULL;
    unsigned int flags;
    size_t bytes;
    float *a, *b, *c;                      // Pinned memory allocated on the CPU
    float *a_UA, *b_UA, *c_UA;             // Non-4K Aligned Pinned memory on the CPU
    float *d_a, *d_b, *d_c;                // Device pointers for mapped memory
    float errorNorm, refNorm, ref, diff;
    hipDeviceProp_t deviceProp;

    if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        printf("Usage:  simpleZeroCopy [OPTION]\n\n");
        printf("Options:\n");
        printf("  --device=[device #]  Specify the device to be used\n");
        printf("  --use_generic_memory (optional) use generic page-aligned for system memory\n");
        return EXIT_SUCCESS;
    }

    /* Get the device selected by the user or default to 0, and then set it. */
    if (getCmdLineArgumentString(argc, (const char **)argv, "device", &device))
    {
        hipGetDeviceCount(&deviceCount);
        idev = atoi(device);

        if (idev >= deviceCount || idev < 0)
        {
            fprintf(stderr, "Device number %d is invalid, will use default CUDA device 0.\n", idev);
            idev = 0;
        }
    }

    // if GPU found supports SM 1.2, then continue, otherwise we exit 
    if (!checkCudaCapabilities(1, 2))
    {
        exit(EXIT_SUCCESS);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "use_generic_memory"))
    {
#if defined(__APPLE__) || defined(MACOSX)
        bPinGenericMemory = false;  // Generic Pinning of System Paged memory is not currently supported on Mac OSX
#else
        bPinGenericMemory = true;
#endif
    }

    if (bPinGenericMemory)
    {
        printf("> Using Generic System Paged Memory (malloc)\n");
    }
    else
    {
        printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
    }

    checkCudaErrors(hipSetDevice(idev));

    /* Verify the selected device supports mapped memory and set the device
       flags for mapping host memory. */

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020

    if (!deviceProp.canMapHostMemory)
    {
        fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);

        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);

    exit(EXIT_SUCCESS);
#endif

#if CUDART_VERSION < 4000

    if (bPinGenericMemory)
    {
        fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);

        exit(EXIT_SUCCESS);
    }

#endif

    /* Allocate mapped CPU memory. */

    nelem = 1048576;
    bytes = nelem*sizeof(float);

    /* dean:
    | 对比项             | malloc + hipHostRegisterMapped        | hipHostMallocMapped        |
    | ------------------ | -------------------------------------- | -------------------------- |
    | 内存分配方式        | 手动 malloc                            | CUDA 自动分配 pinned memory |
    | 是否锁页（Pinned）  | 是（通过 Register 实现）                | 是（由 HostAlloc 默认保证）  |
    | 是否可映射给 device | 是（使用 `hipHostRegisterMapped`）     | 是（由 flag 指定）          |
    | 是否必须对齐        | ✅ 是（需要手动对齐，尤其 > 4K 时）      | ❌ 不用，CUDA 内部已处理    |
    | 是否更灵活          | ✅ 更灵活（可管理原始指针）             | 相对简单但较黑箱             |
    | 推荐平台            | 嵌入式/性能敏感场景（如 Jetson）         | 通用主机平台                |
    | 清理内存            | 需调用 `hipHostUnregister` + `free`   | 只需 `hipHostFree`         |

    在 Jetson AGX Xavier 上的推荐
    因为 Jetson 是 SoC 架构，共享物理内存，因此 Zero-Copy 效果很好。
    malloc + hipHostRegisterMapped 和 hipHostMallocMapped 都是在 Host memory（CPU DRAM） 上分配。
    ✅ Jetson 是 CPU/GPU 共用 DRAM，不像 PC dGPU 要跨 PCIe。
    ✅ GPU 通过 IOMMU 可以直接访问这块 Host Memory（Zero-Copy）。
    建议优先使用 hipHostMallocMapped, 更方便，除非：
        需要手动控制对齐（如 tensor 要求 4K 对齐），
        或已有现成 malloc 指针需要注册成 pinned.
     */
    if (bPinGenericMemory)
    {
        // dean: 手动 malloc，然后注册为 pinned+mapped
#if CUDART_VERSION >= 4000
        a_UA = (float *) malloc(bytes + MEMORY_ALIGNMENT);
        b_UA = (float *) malloc(bytes + MEMORY_ALIGNMENT);
        c_UA = (float *) malloc(bytes + MEMORY_ALIGNMENT);

        // We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
        a = (float *) ALIGN_UP(a_UA, MEMORY_ALIGNMENT);
        b = (float *) ALIGN_UP(b_UA, MEMORY_ALIGNMENT);
        c = (float *) ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

        checkCudaErrors(hipHostRegister(a, bytes, hipHostRegisterMapped));
        checkCudaErrors(hipHostRegister(b, bytes, hipHostRegisterMapped));
        checkCudaErrors(hipHostRegister(c, bytes, hipHostRegisterMapped));
#endif
    }
    else
    {
        // dean: 直接调用 hipHostAlloc 分配 pinned+mapped 内存
#if CUDART_VERSION >= 2020
        flags = hipHostMallocMapped;
        checkCudaErrors(hipHostAlloc((void **)&a, bytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&b, bytes, flags));
        checkCudaErrors(hipHostAlloc((void **)&c, bytes, flags));
#endif
    }

    /* Initialize the vectors. */

    for (n = 0; n < nelem; n++)
    {
        a[n] = rand() / (float)RAND_MAX;
        b[n] = rand() / (float)RAND_MAX;
    }

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */

#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

    /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */
    printf("> vectorAddGPU kernel will add vectors using mapped CPU memory...\n");
    // 4096个block, 每个block 256个thread
    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem/(float)block.x));
    vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("vectorAddGPU() execution failed");

    /* Compare the results */

    printf("> Checking the results from vectorAddGPU() ...\n");
    errorNorm = 0.f;
    refNorm = 0.f;

    for (n = 0; n < nelem; n++)
    {
        ref = a[n] + b[n];
        diff = c[n] - ref;
        errorNorm += diff*diff;
        refNorm += ref*ref;
    }

    errorNorm = (float)sqrt((double)errorNorm);
    refNorm = (float)sqrt((double)refNorm);

    /* Memory clean up */

    printf("> Releasing CPU memory...\n");

    if (bPinGenericMemory)
    {
#if CUDART_VERSION >= 4000
        checkCudaErrors(hipHostUnregister(a));
        checkCudaErrors(hipHostUnregister(b));
        checkCudaErrors(hipHostUnregister(c));
        free(a_UA);
        free(b_UA);
        free(c_UA);
#endif
    }
    else
    {
#if CUDART_VERSION >= 2020
        checkCudaErrors(hipHostFree(a));
        checkCudaErrors(hipHostFree(b));
        checkCudaErrors(hipHostFree(c));
#endif
    }

    exit(errorNorm/refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE);
}
